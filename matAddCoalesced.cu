
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matSum(float* S, float* A, float* B, int N) {
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int tid = i*N + j;

  if (tid < N*N) {
    S[tid] = A[tid] + B[tid];
  }
}


// Fills a vector with random float entries.
void randomInit(float* data, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      int tid = i*N+j;
      data[tid] = (float)drand48();
    }
  }
}


int main(int argc, char* argv[])
{

  if (argc != 2) {
    fprintf(stderr, "Syntax: %s <matrix size> \n", argv[0]);
    return EXIT_FAILURE;
  }


  int N = atoi(argv[1]);

  // set seed for drand48()
  srand48(42);

  // allocate host memory for matrices A and B
  float* A = (float*) malloc(N * N * sizeof(float));
  float* B = (float*) malloc(N * N * sizeof(float));
  float* S = (float*) malloc(N * N * sizeof(float));

  // initialize host matrices
  randomInit(A, N);
  randomInit(B, N);

  // allocate device matrices (linearized)
  float* dev_A = NULL;
  float* dev_B = NULL;
  float* dev_S = NULL;
  checkCuda( hipMalloc((void**) &dev_A, N * N * sizeof(float)) );
  checkCuda( hipMalloc((void**) &dev_B, N * N * sizeof(float)) );
  checkCuda( hipMalloc((void**) &dev_S, N * N * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(dev_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(dev_B, B, N*N*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  int Tile_Width = 8;

  int GridSize = (N + Tile_Width-1) / Tile_Width;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);

  struct timeval t1, t2;
  gettimeofday(&t1, 0);

  matSum<<< gridDim, blockDim >>>(dev_S, dev_A, dev_B, N);
  hipDeviceSynchronize();

  // copy result from device to host
  checkCuda( hipMemcpy( S, dev_S, N * N * sizeof(float),hipMemcpyDeviceToHost) );

  gettimeofday(&t2, 0);
  double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
  printf("%d:%.3f\n", N, time);

  // clean up memory
  free(A);
  free(B);
  free(S);
  checkCuda( hipFree(dev_A) );
  checkCuda( hipFree(dev_B) );
  checkCuda( hipFree(dev_S) );

  return 0;
}

